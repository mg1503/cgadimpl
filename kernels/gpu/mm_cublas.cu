// =========================
// kernels/gpu/mm_cublas.cu
// =========================
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "ad/kernels_api.hpp"

// external linkage (no 'static')
void mm_cuda(const float* A, const float* B, float* C,
             int M, int K, int N, ag_cuda_stream_t s) {
  static thread_local hipblasHandle_t handle = nullptr;
  if (!handle) hipblasCreate(&handle);
  hipblasSetStream(handle, (hipStream_t)s);

  const float alpha = 1.f, beta = 0.f;
  // Row-major trick: C(M,N) = (B^T(N,K) * A^T(K,M))^T
  hipblasSgemm(handle,
              HIPBLAS_OP_T, HIPBLAS_OP_T,
              N, M, K,
              &alpha,
              B, K,   // lda
              A, M,   // ldb
              &beta,
              C, N);  // ldc
}
