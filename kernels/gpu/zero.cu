// =========================
// kernels/gpu/zero.cu
// =========================
#include <hip/hip_runtime.h>
#include <cstdint>
#include "ad/kernels_api.hpp"

// external linkage (no 'static')
void zero_cuda(float* x, int64_t n, ag_cuda_stream_t s) {
  hipMemsetAsync(x, 0, size_t(n) * sizeof(float), (hipStream_t)s);
}
